#include "hip/hip_runtime.h"
/*
 * Author: Liam Lawrence
 * Date: 3.27.17
 *
 * CUDA test code -- cuda function
 */

#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/cuda_devptrs.hpp>

using namespace cv;
using namespace cv::cuda;

__global__ void gpuInRange(const PtrStepSz<uchar3> src, int lh, int ls, int lv, int uh, int us, int uv, PtrStep<uchar3> dst) {

    // Iterates through pixels
	int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // If we haven't run out of pixels yet
    if(x < src.cols && y < src.rows) {
        // Thresholds the three channels
        uchar3 v = src(y,x);
        if(v.z >= lh && v.z <= uh &&
           v.y >= ls && v.y <= us &&
           v.x >= lv && v.x <= uv)
            dst(y,x) = 255; // set to a white pixel
        else
            dst(y,x) = 0;   // set to a black pixel
    }
}

extern "C" void gpuInRange_caller(const PtrStepSz<uchar3>& src, Scalar lowerb, Scalar upperb, PtrStep<uchar3> dst) {
    // Set up memory for iterating through pixels
    dim3 block(32, 8);
    dim3 grid((src.cols + block.x - 1)/block.x,(src.rows + block.y - 1)/block.y);

    /* vvv Test these to see if they are faster vvv
     * const int m = 32;
     * is src.cols or int numCols = src.cols faster if called more than once?
     * const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
     * const dim3 blockSize(m, m, 1);
     */

    // Runs the CUDA function
    gpuInRange<<<grid, block>>>(src, lowerb[0], lowerb[1], lowerb[2], upperb[0], upperb[1], upperb[2], dst);
}
